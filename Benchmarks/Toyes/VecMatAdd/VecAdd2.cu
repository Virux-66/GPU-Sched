/**
 * This is a toy program from CUDA tutorial, it is to compute the addition on
 * two square matrixes, Here I use it to study some basic built-in data
 * structures provided by CUDA
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 4096
#define BLK 32

__global__ void VecAdd(double *A, double *B, double *C) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
  printf("A[%d] (%f) + B[%d] (%f) = C[%d] (%f)\n", i, A[i], i, B[i], i, C[i]);
}

void VecAddWrapper(double *d_A, double *d_B, double *d_C) {
  dim3 ThreadsPerBlock(BLK);
  dim3 BlocksPerGrid(N/BLK);
  VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_A, d_B, d_C);
}

int main(int argc, char **argv) {
  // A and B are input data, C is to store output data
  double A[N], B[N], C[N], sum = 0;
  size_t size = N * sizeof(double);

  // initialize the input data
  for (int i = 0; i < N; i++) {
    A[i] = sin(i) * sin(i);
    B[i] = cos(i) * cos(i);
  }

  // correspoinding memory for data in device
  double *d_src[2], *d_res;
  hipMalloc(&d_src[0], size);
  hipMalloc(&d_src[1], size);
  hipMalloc(&d_res, size);

  printf("Init: %p, %p, %p\n", d_src[0], d_src[1], d_res);

  hipMemcpy(d_src[0], A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_src[1], B, size, hipMemcpyHostToDevice);

  printf("Maturized: %p, %p, %p\n", d_src[0], d_src[1], d_res);

  // cudaMemcpy(d_src[1], d_res, size, cudaMemcpyDeviceToDevice);
  // VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_src[0], d_src[1], d_res);
  VecAddWrapper(d_src[0], d_src[1], d_res);

  hipMemcpy(C, d_res, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    if (i % 64 == 0) printf("C[%d] = %f\n", i, C[i]);
    sum+= C[i];
  }
  printf("sum = %f\n", sum);


  hipFree(d_src[0]);
  hipFree(d_src[1]);
  hipFree(d_res);
  return 0;
}
