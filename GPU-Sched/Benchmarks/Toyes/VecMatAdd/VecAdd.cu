/**
 * This is a toy program from CUDA tutorial, it is to compute the addition on
 * two square matrixes, Here I use it to study some basic built-in data
 * structures provided by CUDA
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 512
#define BLK 32

__global__ void VecAdd(float *A, float *B, float *C) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
  // A and B are input data, C is to store output data
  float A[N], B[N], C[N];
  dim3 ThreadsPerBlock(BLK);
  dim3 BlocksPerGrid(N/BLK);
  size_t size = N * sizeof(float);

  // initialize the input data
  for (int i = 0; i < N; i++) {
    A[i] = sin(i) * sin(i);
    B[i] = cos(i) * cos(i);
  }

  // correspoinding memory for data in device
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipMalloc(&d_A, size * 2);
  hipMalloc(&d_B, size * 2);
  hipMalloc(&d_C, size * 2);

  VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_A, d_B, d_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
