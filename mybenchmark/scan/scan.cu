#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include "scan_common.h"

//All three kernels run 512 threads per workgroup
//Must be a power of two
#define THREADBLOCK_SIZE 256

////////////////////////////////////////////////////////////////////////////////
// Basic scan codelets
////////////////////////////////////////////////////////////////////////////////
//Naive inclusive scan: O(N * log2(N)) operations
//Allocate 2 * 'size' local memory, initialize the first half
//with 'size' zeros avoiding if(pos >= offset) condition evaluation
//and saving instructions
inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size, cg::thread_block cta)
{
    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
    s_Data[pos] = 0;
    pos += size;
    s_Data[pos] = idata;

    for (uint offset = 1; offset < size; offset <<= 1)
    {
        cg::sync(cta);
        uint t = s_Data[pos] + s_Data[pos - offset];
        cg::sync(cta);
        s_Data[pos] = t;
    }

    return s_Data[pos];
}

inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size, cg::thread_block cta)
{
    return scan1Inclusive(idata, s_Data, size, cta) - idata;
}


inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size, cg::thread_block cta)
{
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4, cta);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}

//Exclusive vector scan: the array to be scanned is stored
//in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size, cg::thread_block cta)
{
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size, cta);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Scan kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint size
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, size, cta);

    //Write back
    d_Dst[pos] = odata4;
}

//Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
__global__ void scanExclusiveShared2(
    uint *d_Buf,
    uint *d_Dst,
    uint *d_Src,
    uint N,
    uint arrayLength
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load top elements
    //Convert results of bottom-level scan back to inclusive
    uint idata = 0;

    if (pos < N)
        idata =
            d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] +
            d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];

    //Compute
    uint odata = scan1Exclusive(idata, s_Data, arrayLength, cta);

    //Avoid out-of-bound access
    if (pos < N)
    {
        d_Buf[pos] = odata;
    }
}

//Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
__global__ void uniformUpdate(
    uint4 *d_Data,
    uint *d_Buffer
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint buf;
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0)
    {
        buf = d_Buffer[blockIdx.x];
    }

    cg::sync(cta);

    uint4 data4 = d_Data[pos];
    data4.x += buf;
    data4.y += buf;
    data4.z += buf;
    data4.w += buf;
    d_Data[pos] = data4;
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
//Due to scanExclusiveShared<<<>>>() 1D block addressing
extern "C" const uint MAX_BATCH_ELEMENTS = 64 * 1048576;
extern "C" const uint MIN_SHORT_ARRAY_SIZE = 4;
extern "C" const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;
extern "C" const uint MIN_LARGE_ARRAY_SIZE = 8 * THREADBLOCK_SIZE;
extern "C" const uint MAX_LARGE_ARRAY_SIZE = 4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;

//Internal exclusive scan buffer
//static uint *d_Buf;
/*
extern "C" void initScan(void)
{
    checkCudaErrors(hipMalloc((void **)&d_Buf, (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint)));
}

extern "C" void closeScan(void)
{
    checkCudaErrors(hipFree(d_Buf));
}
*/
static uint factorRadix2(uint &log2L, uint L)
{
    if (!L)
    {
        log2L = 0;
        return 0;
    }
    else
    {
        for (log2L = 0; (L & 1) == 0; L >>= 1, log2L++);

        return L;
    }
}

static uint iDivUp(uint dividend, uint divisor)
{
    return ((dividend % divisor) == 0) ? (dividend / divisor) : (dividend / divisor + 1);
}
/*
extern "C" size_t scanExclusiveShort(
    uint *d_Dst,
    uint *d_Src,
    uint batchSize,
    uint arrayLength
)
{
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);

    //Check supported size range
    assert((arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE));

    //Check total batch size limit
    assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);

    //Check all threadblocks to be fully packed with data
    assert((batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0);

    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength
    );
    getLastCudaError("scanExclusiveShared() execution FAILED\n");

    return THREADBLOCK_SIZE;
}

extern "C" size_t scanExclusiveLarge(
    uint *d_Output,
    uint *d_Input,
    uint batchSize,
    uint arrayLength
)
{
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);

    //Check supported size range
    assert((arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE));

    //Check total batch size limit
    assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);

    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Output,
        (uint4 *)d_Input,
        4 * THREADBLOCK_SIZE
    );
    getLastCudaError("scanExclusiveShared() execution FAILED\n");

    //Not all threadblocks need to be packed with input data:
    //inactive threads of highest threadblock just don't do global reads and writes
    const uint blockCount2 = iDivUp((batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
        (uint *)d_Buf,
        (uint *)d_Output,
        (uint *)d_Input,
        (batchSize *arrayLength) / (4 * THREADBLOCK_SIZE),
        arrayLength / (4 * THREADBLOCK_SIZE)
    );
    getLastCudaError("scanExclusiveShared2() execution FAILED\n");

    uniformUpdate<<<(batchSize *arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Output,
        (uint *)d_Buf
    );
    getLastCudaError("uniformUpdate() execution FAILED\n");

    return THREADBLOCK_SIZE;
}
*/
extern "C" int cuda_main(int argc,char** argv){

    volatile int64_t num_floatingPoint=0;
    volatile int64_t num_transferredBytes=436469760;
    volatile float arithmetic_intensity=0;

    printf("%s Starting...\n\n", argv[0]);

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);

    uint *d_Input, *d_Output, *d_Buf;
    uint *h_Input, *h_OutputCPU, *h_OutputGPU;
    //StopWatchInterface  *hTimer = NULL;
    //const uint N = 13 * 1048576 / 2;
    const uint N = 52 * 1048576;

    printf("Allocating and initializing host arrays...\n");
    h_Input     = (uint *)malloc(N * sizeof(uint));
    h_OutputCPU = (uint *)malloc(N * sizeof(uint));
    h_OutputGPU = (uint *)malloc(N * sizeof(uint));
    srand(2009);

    for (uint i = 0; i < N; i++)
    {
        h_Input[i] = rand();
    }

    printf("Allocating and initializing CUDA arrays...\n");
    hipMalloc((void **)&d_Input, N * sizeof(uint));
    hipMalloc((void **)&d_Output, N * sizeof(uint));
    hipMalloc((void**)&d_Buf,(MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint));
    uint4* d_Output_uint4=(uint4*)d_Output;
    uint4* d_Input_uint4=(uint4*)d_Input;
    hipMemcpy(d_Input, h_Input, N * sizeof(uint), hipMemcpyHostToDevice);

    printf("Initializing CUDA-C scan...\n\n");
    //initScan();

    int globalFlag = 1;
    size_t szWorkgroup;
    const int iCycles = 1;  //replace iCycles by 1, avoiding unnecessary iterations

    printf("***Running GPU scan for large arrays (%u identical iterations)...\n\n", iCycles);

    //uint arrayLength=MIN_LARGE_ARRAY_SIZE;
    uint arrayLength=MAX_LARGE_ARRAY_SIZE;

    //for (uint arrayLength = MIN_LARGE_ARRAY_SIZE; arrayLength <= MAX_LARGE_ARRAY_SIZE; arrayLength <<= 1)
    {
        printf("Running scan for %u elements (%u arrays)...\n", arrayLength, N / arrayLength);
        hipDeviceSynchronize();

        for (int i = 0; i < iCycles; i++)
        {
            //szWorkgroup = scanExclusiveLarge(d_Output, d_Input, N / arrayLength, arrayLength);
            uint batchSize=N / arrayLength;
            uint grid=(batchSize * arrayLength)/(4 * THREADBLOCK_SIZE);
            uint block=THREADBLOCK_SIZE;
            //Check power-of-two factorization
            uint log2L;
            uint factorizationRemainder = factorRadix2(log2L, arrayLength);
            assert(factorizationRemainder == 1);

            //Check supported size range
            assert((arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE));

            //Check total batch size limit
            assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);
            

            scanExclusiveShared<<<grid, block>>>(
                //(uint4 *)d_Output,
                //(uint4 *)d_Input,
                d_Output_uint4,
                d_Input_uint4,
                4 * THREADBLOCK_SIZE
            );
            getLastCudaError("scanExclusiveShared() execution FAILED\n");

            //Not all threadblocks need to be packed with input data:
            //inactive threads of highest threadblock just don't do global reads and writes
            const uint blockCount2 = iDivUp((batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
            scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
                /*(uint *)*/d_Buf,
                /*(uint *)*/d_Output,
                /*(uint *)*/d_Input,
                (batchSize *arrayLength) / (4 * THREADBLOCK_SIZE),
                arrayLength / (4 * THREADBLOCK_SIZE)
            );
            getLastCudaError("scanExclusiveShared2() execution FAILED\n");

            uniformUpdate<<<grid, block>>>(
                //(uint4 *)d_Output,
                d_Output_uint4,
                /*(uint *)*/d_Buf
            );
            getLastCudaError("uniformUpdate() execution FAILED\n");
        }

        hipDeviceSynchronize();

        //printf("Validating the results...\n");
        printf("...reading back GPU results\n");
        hipMemcpy(h_OutputGPU, d_Output, N * sizeof(uint), hipMemcpyDeviceToHost);

        /* 
        printf("...scanExclusiveHost()\n");
        scanExclusiveHost(h_OutputCPU, h_Input, N / arrayLength, arrayLength);

        // Compare GPU results with CPU results and accumulate error for this test
        printf(" ...comparing the results\n");
        int localFlag = 1;

        for (uint i = 0; i < N; i++)
        {
            if (h_OutputCPU[i] != h_OutputGPU[i])
            {
                localFlag = 0;
                break;
            }
        }

        // Log message on individual test result, then accumulate to global flag
        printf(" ...Results %s\n\n", (localFlag == 1) ? "Match" : "DON'T Match !!!");
        globalFlag = globalFlag && localFlag;

        // Data log
        if (arrayLength == MAX_LARGE_ARRAY_SIZE)
        {
            printf("\n");
            //printf("scan, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u Elements, NumDevsUsed = %u, Workgroup = %u\n",
                   //(1.0e-6 * (double)arrayLength/timerValue), timerValue, (unsigned int)arrayLength, 1, (unsigned int)szWorkgroup);
            printf("\n");
        }
        */

    }

    printf("Shutting down...\n");
    //closeScan();
    hipFree(d_Output);
    hipFree(d_Input);
    hipFree(d_Buf);

    free(h_Input);
    free(h_OutputCPU);
    free(h_OutputGPU);

    // pass or fail (cumulative... all tests in the loop)
    //exit(globalFlag ? EXIT_SUCCESS : EXIT_FAILURE);
    return 0;
}